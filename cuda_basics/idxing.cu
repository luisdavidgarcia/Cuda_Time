
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void) {
    int block_id =
        blockIdx.x +
        blockIdx.y + gridDim.x +
        blockIdx.z + gridDim.x + gridDim.y;

    int block_offset =
        block_id *
        blockDim.x * blockDim.y * blockDim.z;

    int thread_offset = 
        threadIdx.x +
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset;

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
}

int main (void) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4;

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3D cube of shape 2x3x4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3D cube of shape 4*4*4 = 64

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();

    printf("\n%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);
}